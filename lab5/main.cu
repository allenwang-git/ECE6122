#include "hip/hip_runtime.h"
/*
Author: Yinuo Wang
Class: ECE 6122
Last Date Modified: 11/13/2021

Description:
This program can simulate the temperature diffusing process of a plate.
User neesd to specify the interior points dimension and iteration times of the simulation.
The program will check the user input and print message if any invalid arguments are input.
The final temperature matrix will be ouput to the "finalTemperatures.csv".

Reference:
[NVIDIA CUDA sample programs](https://github.com/NVIDIA/cuda-samples)

*/


#include <hip/hip_runtime.h>
#include "main.h"

/*
 * This function is the gpu kernel function to update the temperature
 * @param H is the temperature matrix of points
 * @param N is the matrix dimension
 */
 __global__ void updateTemperatureCUDA(double *H, int N)
{
    // generate index
    int tID_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tID_y = blockIdx.y * blockDim.y + threadIdx.y;

    double currentTemperature = 0.;
    // update the temperature
    if (tID_x >= 1 && tID_x <= (N-2) && tID_y >= 1 && tID_y <= (N-2))
        currentTemperature = 0.25 *(H[(tID_x+1)+tID_y*N] + H[(tID_x-1)+tID_y*N] + H[tID_x+(tID_y-1)*N] + H[tID_x+(tID_y+1)*N]);
    else
        currentTemperature = H[tID_x+tID_y*N];

    H[tID_x + tID_y*N] = currentTemperature;

    __syncthreads();
}

/*
 * This function is used to simulate the temperature diffusion process
 * @param N is the points in the plate
 * @param iteration is the total updating iteration of the simulation
 * */
void computeTemperature(const int &N, const long iteration)
{
    //Alloacte host memory
    unsigned long mem_size = sizeof(double)* N * N;
    double *H;
    hipMallocManaged((void**)&H, sizeof(double) * mem_size);
    if (H == NULL)
    {
        fprintf(stderr, "Failed to allocate memory!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the temperature
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if(i==0&&j<0.7*(N-1)&&j>0.3*(N-1))
                H[i*N+j] = 100.0;

            else
                H[i*N+j] = 20.0;
        }
    }

    hipError_t error;

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Do the iteration
    dim3 blocks(N,1);
    dim3 threads(1,N);
    for (int iter = 0; iter < iteration; ++iter)
    {
        // call gpu cuda function
        updateTemperatureCUDA<<<blocks, threads>>>(H, N);
    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    printf("Cuda Kernel Execute Time: %.2f msec\n", msecTotal);

    // Synchronize between device and host
    hipDeviceSynchronize();

    //   Create a file for output result
    fstream outputFile("finalTemperatures.csv", ios::out | ios::trunc);
    if (outputFile.good())
    {
        for (int i = 0; i < N; ++i)
        {
            for (int j = 0; j < N; ++j)
            {
                outputFile <<H[j+i*N] <<",";
            }
            outputFile<<endl;
        }
    }
    else
    {
        cout << "[ECE6122-Lab5] Can not open output file.\n";
    }

}

/*
 *  Main function to check the imput and call the temperature update function
 */
int main(int argc, char **argv)
{
    printf("[2D Steady State Heat Conduction]\n");

    // Set the point number and iteration as what is provided at the command line
    int N = 100, iteration=1000, argValue, argIndex;

    // check -N argument
    if (checkCmdLineArg(argc, (const char **)argv, "-N", argIndex))
    {
        if(argIndex+1 < argc && isValidArg(argv[argIndex+1],argValue))
        {
            N = argValue + 2;
        }
        else
        {
            printf("Invalid parameters, please check your values for '-N'.\n");
            exit(EXIT_FAILURE);
        }
    }
    else
    {
        N = N + 2;
    }
    // check -I argument
    if (checkCmdLineArg(argc, (const char **)argv, "-I", argIndex))
    {
        if(argIndex+1 < argc && isValidArg(argv[argIndex+1],argValue))
        {
            iteration = argValue;
        }
        else
        {
            printf("Invalid parameters,please check your values for '-I'.\n");
            exit(EXIT_FAILURE);
        }
    }

    // check device properties
    int devID = 0;
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDeviceProperties(&deviceProp, devID);
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device: \"%s\" \n", deviceProp.name);
    }

    cout<<"Total Points: "<<N<<"*"<<N <<" Iteration times: "<<iteration<<endl;

    // Beigin to compute the temperature
    computeTemperature(N, iteration);

    return 0;
}